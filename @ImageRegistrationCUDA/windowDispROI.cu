
#include <hip/hip_runtime.h>
__global__ void windowROI (double2 *roi_1, double2 *roi_2, const double2 *padded_template,
								const double2 *padded_target, const double *window2D, const int *points,
								const int *dispX, const int *dispY, const int HalfCCSize, const int PointLength, const int ImageWidth, const int ImageHeight) {

	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int i_ = blockIdx.z * blockDim.z + threadIdx.z;
	
	int CCSize = (2 * HalfCCSize) + 1;
	
	if (i_ >= PointLength)
		return;
	
	int x1 = points[i_] - HalfCCSize - 1 + j;
	int y1 = points[i_ + PointLength] - HalfCCSize - 1 + i;
	int x2 = points[i_] - HalfCCSize - 1 + j + dispX[i_];
	int y2 = points[i_ + PointLength] - HalfCCSize - 1 + i + dispY[i_];
	
	if (i >= CCSize || j >= CCSize)
		return;
	
	double2 r1, r2;
	
	double2 I1 = padded_template[x1*ImageHeight + y1];
	double2 I2 = padded_target[x2*ImageHeight + y2];
	
	r1.x = window2D[j*CCSize + i] * I1.x;
	r1.y = window2D[j*CCSize + i] * I1.y;
	r2.x = window2D[j*CCSize + i] * I2.x;
	r2.y = window2D[j*CCSize + i] * I2.y;
	
	int indx = (i_ * CCSize * CCSize) + (CCSize * j) + i; //(i * CCSize * PointLength) + (j * PointLength) + i_;
	roi_1[indx] = r1;
	roi_2[indx] = r2;
}