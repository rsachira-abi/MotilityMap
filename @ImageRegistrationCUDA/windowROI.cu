
#include <hip/hip_runtime.h>
__global__ void windowROI (double2 *roi_1, double2 *roi_2, const double2 *padded_template,
								const double2 *padded_target, const double *window2D, const int *points,
								const int HalfCCSize,const int PointLength, const int ImageWidth, const int ImageHeight) {

	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int i_ = blockIdx.z * blockDim.z + threadIdx.z;
	
	int CCSize = (2 * HalfCCSize) + 1;
	
	if (i_ >= PointLength)
		return;
	
	int x = points[i_] - HalfCCSize - 1 + j;
	int y = points[i_ + PointLength] - HalfCCSize - 1 + i;
	
	int x_end = points[i_] + HalfCCSize;
	int y_end = points[i_ + PointLength] + HalfCCSize;
	
	if (x >= x_end || y >= y_end)
		return;
	
	double2 r1, r2;
	
	double2 I1 = padded_template[x*ImageHeight + y];
	double2 I2 = padded_target[x*ImageHeight + y];
	
	r1.x = window2D[j*CCSize + i] * I1.x;
	r1.y = window2D[j*CCSize + i] * I1.y;
	r2.x = window2D[j*CCSize + i] * I2.x;
	r2.y = window2D[j*CCSize + i] * I2.y;
	
	int indx = (i_ * CCSize * CCSize) + (CCSize * j) + i; //(i * CCSize * PointLength) + (j * PointLength) + i_;
	roi_1[indx] = r1;
	roi_2[indx] = r2;
}