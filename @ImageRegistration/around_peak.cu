
#include <hip/hip_runtime.h>
__global__ void around_peak (double *CC, const int *maxX, const int *maxY, const int SquareSize, const double peakThreshold, const int HalfCCSize, const int PointLength) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int i_ = blockIdx.z * blockDim.z + threadIdx.z;
	
	int CCSize = (2 * HalfCCSize) + 1;
	
	if (i_ >= PointLength || i >= CCSize || j >= CCSize)
		return;
	
	int j_min = maxX[i_] - SquareSize;
	int i_min = maxY[i_] - SquareSize;
	int j_max = maxX[i_] + SquareSize;
	int i_max = maxY[i_] + SquareSize;
	
	int indx = (i_ * CCSize * CCSize) + (j * CCSize) + i;
	
	if (i >= i_min && i <= i_max && j >= j_min && j <= j_max) {
		
		double CCval = CC[indx];
		
		if (CCval > peakThreshold)
			CC[indx] = 1;
		else
			CC[indx] = 0;
	}
	else {
		CC[indx] = 0;
	}
}